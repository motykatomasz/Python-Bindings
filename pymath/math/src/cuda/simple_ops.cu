#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <math/cuda_ops.hpp>

__global__ void square(int* d_out, const int base){
    *d_out = base * base;
}

__global__ void cube(int* d_out, const int base){
    *d_out = base * base * base;
}

int square_cuda(int base) {
    int BYTES = sizeof(int);

    // Declare pointer to cude in and out memory
    int h_out;
    int* d_out;

    // Allocate memory on the GPU
    hipMalloc((void**) &d_out, BYTES);

    // Start kernel
    square<<<1, 1>>>(d_out, base);

    // Copy the output data from device to host
    hipMemcpy(&h_out, d_out, BYTES, hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_out);
    
    return h_out;
}

int cube_cuda(int base) {
    int BYTES = sizeof(int);

    // Declare pointer to cude in and out memory
    int h_out;
    int* d_out;

    // Allocate memory on the GPU
    hipMalloc((void**) &d_out, BYTES);

    // Start kernel
    cube<<<1, 1>>>(d_out, base);

    // Copy the output data from device to host
    hipMemcpy(&h_out, d_out, BYTES, hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_out);
    
    return h_out;
}